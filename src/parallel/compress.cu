#include "hip/hip_runtime.h"
#include "../../include/parallel.hpp"

#include "../../include/stb_image.h"
#include "../../include/stb_image_write.h"

#include <hip/hip_runtime.h>

__global__ void downScaleKernel(unsigned char* inputImage, unsigned char* outputImage, int width, int height, int channels, int outWidth, int outHeight, int scaleFactor) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < outWidth && y < outHeight) {
        int inputIdx = (y * scaleFactor * width + x * scaleFactor) * channels;
        int outputIdx = (y * outWidth + x) * channels;

        for (int c = 0; c < channels; ++c) {
            outputImage[outputIdx + c] = inputImage[inputIdx + c];
        }
    }
}


void compressImageLossyCUDA(unsigned char* inputImage, unsigned char* outputImage, int width, int height, int channels, int outWidth, int outHeight, int scaleFactor) {
    unsigned char *d_inputImage, *d_outputImage;
    
    hipMalloc((void**)&d_inputImage, width * height * channels * sizeof(unsigned char));
    hipMalloc((void**)&d_outputImage, outWidth * outHeight * channels * sizeof(unsigned char));

    hipMemcpy(d_inputImage, inputImage, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((outWidth + blockSize.x - 1) / blockSize.x, (outHeight + blockSize.y - 1) / blockSize.y);
    downScaleKernel<<<gridSize, blockSize>>>(d_inputImage, d_outputImage, width, height, channels, outWidth, outHeight, scaleFactor);
    hipDeviceSynchronize();
    hipMemcpy(outputImage, d_outputImage, outWidth * outHeight * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}